#include "hip/hip_runtime.h"
#include "db/common.h"
#include "db/cholesky_band_cuda.h"

void cholesky_band_parallel_cuda( 
        const BandMatrix & A, 
        BandMatrix & L, 
        BandMatrix & D ) 
{
    //  init D and L with entries in A
    for( dim_t i = 0; i < A._matDim; i++ ) {
        D.writeEntry( i, i, A.getEntry( i, i ) );
        L.writeEntry( i, i, 1 );
        for( dim_t j = 1; j <= A._lowerBand && (i+j < A._matDim); j++ ) {
            L.writeEntry( i+j, i, A.getEntry( i+j, i ) );
        }
    }

    //  Copy A to device global memory
    const data_t * const hostA = &A._vals[0];
    data_t * hostD = &D._vals[0];
    data_t * hostL = &L._vals[0];
    data_t * devA = NULL;
    data_t * devD = NULL;
    data_t * devL = NULL;

    //  allocate memory in device
    myCudaCheck( hipMalloc( (void**)&devA, sizeof(data_t) * A.getNumNonZeroEntries() ) );
    myCudaCheck( hipMalloc( (void**)&devD, sizeof(data_t) * D.getNumNonZeroEntries() ) );
    myCudaCheck( hipMalloc( (void**)&devL, sizeof(data_t) * L.getNumNonZeroEntries() ) );
    myCudaCheck( hipMemcpy( devA, hostA, sizeof(data_t) * A.getNumNonZeroEntries(), hipMemcpyHostToDevice ) );
    myCudaCheck( hipMemcpy( devD, hostD, sizeof(data_t) * D.getNumNonZeroEntries(), hipMemcpyHostToDevice ) );
    myCudaCheck( hipMemcpy( devL, hostL, sizeof(data_t) * L.getNumNonZeroEntries(), hipMemcpyHostToDevice ) );

    //  solve column by column
    for( dim_t colIdx = 0; colIdx < A._matDim; colIdx ++ ) {
        choleskyColumnSolverKernel<<< 1, L._lowerBand+1, (2*L._lowerBand+1)*sizeof(data_t)>>>( devA, devD, devL, colIdx, L._matDim, L._lowerBand );
    }

    myCudaCheck( hipMemcpy( hostD, devD, sizeof(data_t) * D.getNumNonZeroEntries(), hipMemcpyDeviceToHost ) );
    myCudaCheck( hipMemcpy( hostL, devL, sizeof(data_t) * L.getNumNonZeroEntries(), hipMemcpyDeviceToHost ) );
        
    std::cout << "cholesky on band matrix finishes... [parallel version (cuda)]\n";
}

__global__ void choleskyColumnSolverKernel( data_t * devA, data_t * devD, data_t * devL, const dim_t colIdx, const dim_t matDim, const dim_t bandWidth ) 
{
    if( colIdx + threadIdx.x >= matDim ) {
        return;
    }
    extern __shared__ data_t temp[];
    data_t * prevD = &temp[0];          //  D[(col-k):(col-1)]
    data_t * currD = &temp[bandWidth];  //  D[col]
    data_t * prevL = &temp[bandWidth+1];//  L[col, (col-k):(col-1)]    

    if( threadIdx.x == 0 ) {
        //  devD -> prevD, currD
        for( dim_t i = colIdx-1; (i >= colIdx-bandWidth) && i>=0; i-- ) {
            prevD[ i - (colIdx-bandWidth) ] = devD[i];
        }
        currD[0] = devD[colIdx];
    } else {
        //  devL -> prevL
        dim_t row = colIdx;
        dim_t col = colIdx-threadIdx.x;
        if( col >= 0 ) {
            prevL[col - (colIdx-bandWidth)] = devL[col*(bandWidth+1) + (row-col)];
        }
    };
    __syncthreads();

    dim_t col = colIdx;
    dim_t row = colIdx + threadIdx.x;
    data_t currL = 0;
    if( threadIdx.x == 0 ) {
        for( dim_t i = col-1; (i >= col-bandWidth) && i>=0; i-- ) {
            currD[0] -= prevL[i-(col-bandWidth)]*prevL[i-(col-bandWidth)]*prevD[i-(col-bandWidth)];
        }
    } else {
        if( row < matDim ) {
            currL = devL[col*(bandWidth+1) + (row-col)];
            for( dim_t i = col-1; (i >= col-bandWidth+threadIdx.x) && i>=0; i-- ) {
                currL -= prevD[i-(col-bandWidth)]*prevL[i-(col-bandWidth)]*devL[i*(bandWidth+1) + (row-i)];
            }
        }
    }

    __syncthreads();
    if( threadIdx.x == 0 ) {
        devD[colIdx] = currD[0];
    } else {
        if( row < matDim ) {
            devL[col*(bandWidth+1) + (row-col)] = currL / currD[0];
        }
    }
}
